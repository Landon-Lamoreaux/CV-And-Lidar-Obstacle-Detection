#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 128

__global__ void av3(int n, float *in1, float *in2, float *in3, float *out)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// Guarantees that index does not go beyond vector size and applies average
	if (index<n)
	{
			out[index] = (in1[index] + in2[index] + in3[index])/3;
	}
}

float *average3(int num, float *in1, float *in2, float *in3, float *out) 
{
	// Device copies of three inputs and output, size of allocated memory, num of threads and blocks
	float *d_in1, *d_in2, *d_in3, *d_out; 
	int size = num * sizeof(float);
	int thr, blk;
	// Alloc memory for device copies of inputs and outputs
	hipMalloc((void **)&d_in1, size);
	hipMalloc((void **)&d_in2, size);
	hipMalloc((void **)&d_in3, size);
	hipMalloc((void **)&d_out, size);
	// Copy inputs to device
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in3, in3, size, hipMemcpyHostToDevice);
	// Calculates blocks and threads and launch average3 kernel on GPU
	blk=floor(num/THREADS_PER_BLOCK)+1;
	thr=THREADS_PER_BLOCK;
	av3<<<blk,thr>>>(num, d_in1, d_in2, d_in3, d_out);
	// Wait for the GPU to finish
	hipDeviceSynchronize();
	// Copy result back to host and cleanup
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	hipFree(d_in1); hipFree(d_in2); hipFree(d_in3); hipFree(d_out);
	return out;
}